#include "hip/hip_runtime.h"
/*
 * This is a CUDA code that performs an iterative reverse edge 
 * detection algorithm.
 *
 * Training material developed by James Perry and Alan Gray
 * Copyright EPCC, The University of Edinburgh, 2010 
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <sys/types.h>
#include <sys/time.h>


/* Forward Declarations of utility functions*/
double get_current_time();
void datread(char*, void*, int, int);
void pgmwrite(char*, void*, int, int);
void checkCUDAError(const char*);


/* Dimensions of image */
#define WIDTH 256
#define HEIGHT 192

/* Number of iterations to run */
#define ITERATIONS 10

/* Dimensions of CUDA grid and block */
#define GRID_W  16
#define GRID_H  12
#define BLOCK_W 16
#define BLOCK_H 16

/* Maximum difference allowed between host result and GPU result */
#define MAX_DIFF 0.01

/* Data buffer to read edge data into */
float edge[HEIGHT][WIDTH];

/* Data buffer for the resulting image */
float img[HEIGHT][WIDTH];

/* Work buffers, with halos */
float host_input[HEIGHT+2][WIDTH+2];
float gpu_output[HEIGHT+2][WIDTH+2];
float host_output[HEIGHT+2][WIDTH+2];


/* The actual CUDA kernel that runs on the GPU - 1D version by column */
__global__ void inverseEdgeDetect1D_col(float *d_output, float *d_input, float *d_edge)
{
  int col, row;
  int idx, idx_south, idx_north, idx_west, idx_east;
  int numcols = WIDTH + 2;

  /*
   * calculate global row index for this thread  
   * from blockIdx.x, blockDim.x and threadIdx.x
   * remember to add 1 to account for halo    
   */
  row = blockIdx.x*blockDim.x + threadIdx.x + 1;

  /*
   * loop over all columns of the image
   */
  for (col = 1; col <= WIDTH; col++) {
      /*
       * calculate linear index from col and row, for the centre
       * and neighbouring points needed below.
       * For the neighbouring points you need to add/subtract 1  
       * to/from the row or col indices.
       */
      
      idx = row * numcols + col;
      idx_south = (row - 1) * numcols + col;
      idx_north = (row + 1) * numcols + col;
      
      idx_west = row * numcols + (col - 1);
      idx_east = row * numcols + (col + 1);
      
      /* perform stencil operation */  
      d_output[idx] = (d_input[idx_south] + d_input[idx_west] \
		       + d_input[idx_north] + d_input[idx_east] \
		       - d_edge[idx]) * 0.25;

    }
}

/* The actual CUDA kernel that runs on the GPU - 1D version by row */
__global__ void inverseEdgeDetect1D_row(float *d_output, float *d_input, \
					float *d_edge)
{
  int col, row;
  int idx, idx_south, idx_north, idx_west, idx_east;
  int numcols = WIDTH + 2;

  /*
   * calculate global column index for this thread  
   * from blockIdx.x,blockDim.x and threadIdx.x    
   * remember to add 1 to account for halo     
   */
  // col = ;

  /*
   * loop over all rows of the image
   */
  // for ( ; ; )
  {
      /*
       * calculate linear index from col and row, for the centre
       * and neighbouring points needed below.
       * For the neighbouring points you need to add/subtract 1  
       * to/from the row or col indices.
       */      
      idx = row * numcols + col;
      idx_south = (row - 1) * numcols + col;
      idx_north = (row + 1) * numcols + col;
      
      idx_west = row * numcols + (col - 1);
      idx_east = row * numcols + (col + 1);
      
      /* perform stencil operation */  
      d_output[idx] = (d_input[idx_south] + d_input[idx_west] + \
		       d_input[idx_north] + d_input[idx_east] - \
		       d_edge[idx]) * 0.25;
  }
}



/* The actual CUDA kernel that runs on the GPU - 2D version */
__global__ void inverseEdgeDetect2D(float *d_output, float *d_input, \
				    float *d_edge)
{
  int col, row;
  int idx, idx_south, idx_north, idx_west, idx_east;
  int numcols = WIDTH + 2;

  /*
   * calculate global column index for this thread  
   * from blockIdx.x,blockDim.x and threadIdx.x    
   * remember to add 1 to account for halo     
   */
  //col = ;

  /*
   * calculate global row index for this thread  
   * from blockIdx.y,blockDim.y and threadIdx.y
   * remember to add 1 to account for halo    
   */
  //row = ;
    

  /*
   * calculate linear index from col and row, for the centre
   * and neighbouring points needed below.
   * For the neighbouring points you need to add/subtract 1  
   * to/from the row or col indices.
   */
  idx = row * numcols + col;
  idx_south = (row - 1) * numcols + col;
  idx_north = (row + 1) * numcols + col;

  idx_west = row * numcols + (col - 1);
  idx_east = row * numcols + (col + 1);

  /* perform stencil operation */
  d_output[idx] = (d_input[idx_south] + d_input[idx_west] + d_input[idx_north]
              + d_input[idx_east] - d_edge[idx]) * 0.25;
}

int main(int argc, char *argv[])
{
  int x, y;
  int i;
  int errors;

  double start_time_inc_data, end_time_inc_data;
  double cpu_start_time, cpu_end_time;

  float *d_input, *d_output, *d_edge, *tmp;

  size_t memSize = (WIDTH+2) * (HEIGHT+2) * sizeof(float);

  printf("Grid size: %dx%d\n", GRID_W, GRID_H);
  printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

  /* allocate memory on device */
  hipMalloc((void**)&d_input, memSize);
  hipMalloc((void**)&d_output, memSize);
  hipMalloc((void**)&d_edge, memSize);

  /* read in edge data */
  datread("edge256x192.dat", (void *)edge, WIDTH, HEIGHT);

  /* zero buffer so that halo is zeroed */
  for (y = 0; y < HEIGHT+2; y++) {
    for (x = 0; x < WIDTH+2; x++) {
      host_input[y][x] = 0.0;
    }
  }

  /* copy input to buffer with halo */
  for (y = 0; y < HEIGHT; y++) {
    for (x = 0; x < WIDTH; x++) {
      host_input[y+1][x+1] = edge[y][x];
    }
  }

  /*
   * copy to all the GPU arrays. d_output doesn't need to have this data but
   * this will zero its halo
   */
  start_time_inc_data = get_current_time();
  hipMemcpy( d_input, (void *)host_input, memSize, hipMemcpyHostToDevice);
  hipMemcpy( d_output, (void *)host_input, memSize, hipMemcpyHostToDevice);
  hipMemcpy( d_edge, (void *)host_input, memSize, hipMemcpyHostToDevice);

  /* run on GPU */
  for (i = 0; i < ITERATIONS; i++) {

    /* run the kernel */
    dim3 dimGrid(GRID_H);
    dim3 dimBlock(BLOCK_H);
    inverseEdgeDetect1D_col<<< dimGrid, dimBlock >>>(d_output, d_input, d_edge);

    hipDeviceSynchronize();

    /* copy the data back from the output buffer on the device */
    hipMemcpy((void *)gpu_output, d_output, memSize, hipMemcpyDeviceToHost);

    /* copy the new data to the input buffer on the device */
    hipMemcpy( d_input, (void *)gpu_output, memSize, hipMemcpyHostToDevice);
  }

  end_time_inc_data = get_current_time();

  /*
   * run on host for comparison
   */
  cpu_start_time = get_current_time();
  for (i = 0; i < ITERATIONS; i++) {

    /* perform stencil operation */
    for (y = 0; y < HEIGHT; y++) {
      for (x = 0; x < WIDTH; x++) {
	host_output[y+1][x+1] = (host_input[y+1][x] + host_input[y+1][x+2] + \
				 host_input[y][x+1] + host_input[y+2][x+1] \
				 - edge[y][x]) * 0.25;
      }
    }
    
    /* copy output back to input buffer */
    for (y = 0; y < HEIGHT; y++) {
      for (x = 0; x < WIDTH; x++) {
	host_input[y+1][x+1] = host_output[y+1][x+1];
      }
    }
  }
  cpu_end_time = get_current_time();

  /* check that GPU result matches host result */
  errors = 0;
  for (y = 0; y < HEIGHT; y++) {
    for (x = 0; x < WIDTH; x++) {
      float diff = fabs(gpu_output[y+1][x+1] - host_output[y+1][x+1]);
      if (diff >= MAX_DIFF) {
        errors++;
        printf("Error at %d,%d (CPU=%f, GPU=%f)\n", x, y, \
	       host_output[y+1][x+1], \
               gpu_output[y+1][x+1]);
      }
    }
  }
  if (errors == 0) printf("Correct\n");

  /* copy result to output buffer */
  for (y = 0; y < HEIGHT; y++) {
    for (x = 0; x < WIDTH; x++) {
      img[y][x] = gpu_output[y+1][x+1];
    }
  }

  /* write PGM */
  pgmwrite("output.pgm", (void *)img, WIDTH, HEIGHT);

  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_edge);

  printf("GPU Time (Including Data Transfer): %fs\n", \
	 end_time_inc_data - start_time_inc_data);
  printf("CPU Time                          : %fs\n", \
	 cpu_end_time - cpu_start_time);

  return 0;
}


/* Utility Functions */

/*
 * Function to get an accurate time reading
 */
double get_current_time()
{
   static int start = 0, startu = 0;
   struct timeval tval;
   double result;

   if (gettimeofday(&tval, NULL) == -1)
      result = -1.0;
   else if(!start) {
      start = tval.tv_sec;
      startu = tval.tv_usec;
      result = 0.0;
   }
   else
      result = (double) (tval.tv_sec - start) + 1.0e-6*(tval.tv_usec - startu);

   return result;
}


/* Read the input file containing the edge data */
void datread(char *filename, void *vx, int nx, int ny)
{ 
  FILE *fp;

  int nxt, nyt, i, j, t;

  float *x = (float *) vx;

  if (NULL == (fp = fopen(filename,"r")))
  {
    fprintf(stderr, "datread: cannot open <%s>\n", filename);
    exit(-1);
  }

  fscanf(fp,"%d %d",&nxt,&nyt);

  if (nx != nxt || ny != nyt)
  {
    fprintf(stderr,
            "datread: size mismatch, (nx,ny) = (%d,%d) expected (%d,%d)\n",
            nxt, nyt, nx, ny);
    exit(-1);
  }

  for (j=0; j<ny; j++)
  {
    for (i=0; i<nx; i++)
    {
      fscanf(fp,"%d", &t);
      x[(ny-j-1)*nx + i] = t;
    }
  }

  fclose(fp);
}

/* Write the output image as a PGM file */
void pgmwrite(char *filename, void *vx, int nx, int ny)
{
  FILE *fp;

  int i, j, k, grey;

  float xmin, xmax, tmp;
  float thresh = 255.0;

  float *x = (float *) vx;

  if (NULL == (fp = fopen(filename,"w")))
  {
    fprintf(stderr, "pgmwrite: cannot create <%s>\n", filename);
    exit(-1);
  }

  /*
   *  Find the max and min absolute values of the array
   */

  xmin = fabs(x[0]);
  xmax = fabs(x[0]);

  for (i=0; i < nx*ny; i++)
  {
    if (fabs(x[i]) < xmin) xmin = fabs(x[i]);
    if (fabs(x[i]) > xmax) xmax = fabs(x[i]);
  }

  fprintf(fp, "P2\n");
  fprintf(fp, "# Written by pgmwrite\n");
  fprintf(fp, "%d %d\n", nx, ny);
  fprintf(fp, "%d\n", (int) thresh);

  k = 0;

  for (j=ny-1; j >=0 ; j--)
  {
    for (i=0; i < nx; i++)
    {
      /*
       *  Access the value of x[i][j]
       */

      tmp = x[j*nx+i];

      /*
       *  Scale the value appropriately so it lies between 0 and thresh
       */

      if (xmin < 0 || xmax > thresh)
      {
        tmp = (int) ((thresh*((fabs(tmp-xmin))/(xmax-xmin))) + 0.5);
      }
      else
      {
        tmp = (int) (fabs(tmp) + 0.5);
      }

      /*
       *  Increase the contrast by boosting the lower values
       */
     
      grey = (int) (thresh * sqrt(tmp/thresh));

      fprintf(fp, "%3d ", grey);

      if (0 == (k+1)%16) fprintf(fp, "\n");

      k++;
    }
  }

  if (0 != k%16) fprintf(fp, "\n");
  fclose(fp);
}

/* Simple utility function to check for CUDA runtime errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

